
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <sstream>
#include <string>

// export PATH=$PATH:/usr/local/cuda/bin

// CUDA kernel function to set 1 to all values in the float array
__global__
void setOnes(float * my_array) {

  int array_idx = blockIdx.x;
  // int tmp = threadIdx.x;

  my_array[array_idx] = 1;
}

int main(int argc, char * argv[]) {

    float * cpu_zero_array = new float[100];
    memset(cpu_zero_array, 0, sizeof(float) * 100);

    float * gpu_zero_array;
    hipMalloc(&gpu_zero_array, 100 * sizeof(float));
    hipMemcpy(gpu_zero_array, cpu_zero_array, 100 * sizeof(float), hipMemcpyHostToDevice);
    setOnes<<<100,1>>>(gpu_zero_array);
    hipMemcpy(cpu_zero_array, gpu_zero_array, 100 * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < 100; ++i)
        std::cout << cpu_zero_array[i] << std::endl;

}

























