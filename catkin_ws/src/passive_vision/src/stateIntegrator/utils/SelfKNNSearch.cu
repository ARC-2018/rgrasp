
#include <hip/hip_runtime.h>
__global__ void SelfKNNSearch( int * result, const int * args, const float * pc1)
{
    
    int cudaNumBlocks = args[0];
    int cudaNumThreads = args[1];
    int K = args[2];
    int pc1NumPts = args[3];

    int pc1Idx = blockIdx.x * cudaNumThreads + threadIdx.x;

    // the point of reference
    
    float currPtX = pc1[pc1Idx * 3 + 0];
    float currPtY = pc1[pc1Idx * 3 + 1];
    float currPtZ = pc1[pc1Idx * 3 + 2];

    //float* nn_dists;  // a list of distances from low to high
    //int* nn_index;  // a list of indices corresponding to nn_dists
    float nn_dists[15];   
    int nn_index[15];
    
    float inf = 100000.0f;
    
    int w = 20;
    if (pc1Idx < pc1NumPts) {
      for(int i = 0; i < K; i++){
        nn_dists[i] = inf;
        nn_index[i] = pc1Idx;
      }
      
      //nn_dists = (float*)malloc(K * sizeof(float));
      //nn_index = (int*)malloc(K * sizeof(int));
      
      
      nn_dists[0] = 0.0;
      nn_index[0] = pc1Idx;
      
      //for(int dx = max(-w, 0); dx <= min(w, 480); dx++){
        //for(int dy = max(-w, 0); dy <= min(w, 640); dy++){
          
        for (int i = 0; i < pc1NumPts/* && i<60*/; i++) {
          
          float otherPtX = pc1[i * 3 + 0]; 
          float otherPtY = pc1[i * 3 + 1];
          float otherPtZ = pc1[i * 3 + 2];
          float dx = (currPtX - otherPtX), dy = (currPtY - otherPtY), dz = (currPtZ - otherPtZ);
          float dist = dx*dx + dy*dy + dz*dz;

          // find a place to insert
          int insert_idx = 0;
          if(i == pc1Idx){  // will be the first one
            continue;
          }
          else if(dist > 0.1*0.1 || dist > nn_dists[K-1]){  // dist too far or worse than the worst one in the current K list 
            // skip the point
            continue;
          }
          else{
            for(insert_idx = 0; insert_idx < K; insert_idx++){
              if(dist < nn_dists[insert_idx]){
                break;
              }
            }
          }
          
          // if found a place
          if(insert_idx < K){
            //~ // move the things after insert_idx back
            for(int j = K-1; j > insert_idx; j--){
              nn_dists[j] = nn_dists[j-1];
              nn_index[j] = nn_index[j-1];
            }
              
            //~ // save the result to the idx
            nn_dists[insert_idx] = dist;
            nn_index[insert_idx] = i;
          }
        }
      // copy the result
      
      for(int i = 0; i < K; i++){
        result[pc1Idx * K + i] = nn_index[i];  // later need to +1 get to matlab index  // the result will becomes K * pc1NumPts
      }
    }
}
